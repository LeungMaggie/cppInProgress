
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>

__global__ void vec_add(float* a, float* b, float* c)
{
  int i = threadIdx.x;
  c[i] = a[i] + b[i];
}

// int main()
// {
//   std::vector<float> a, b, c;
//   int n = 100;
//   vec_add<<<1, n>>>(a.data(), b.data(), c.data());    // <<<nb blocks per grid, nb threads per block>>>
// }

/////////////////////////////////
constexpr int N = 128;

__global__ void mat_add(float a[N][N], float b[N][N], float c[N][N])
{
  int i = threadIdx.x;
  int j = threadIdx.y;
  c[i][j] = a[i][j] + b[i][j];
}

int main()
{
  // one block of N * N * 1 threads
  int numBlocks = 1;
  dim3 threadsPerBlock(N, N);
  // mat_add<<<numBlocks, threadsPerBlock>>>(a, b, c);
}
