
#include <hip/hip_runtime.h>
/*
Youtube channel: Tom Nurkkala
Intro to GPU Programming
*/

// GPU code
__global__ void square(float* d_out, float* d_in)
{
  int idx = threadIdx.x;      // threadIdx.x : from 0 to number of threads
  float f = d_in[idx];
  d_out[idx] = f * f;
}

// CPU code
int main(int argc, char** argv)
{
  const int arraySize = 64;
  const int arrayBytes = arraySize * sizeof(float);

  // declare and initialize CPU arrays
  float h_in[arraySize];    // h_ stands for host
  float h_out[arraySize];
  for (int i = 0; i < arraySize; ++i)
  {
    h_in[i] = float(i);
  }

  // declare and allocate GPU memory
  float* d_in;              // d_ stands for device
  float* d_out;
  /**
   * We don't get a pointer back from cudaMalloc that refers to that memory on the GPU.
   * Instead, we pass the address of a variable, that's gonna be filled in by cudaMalloc
   * with information that we can use to refer to the memory on the graphics card.
   * Notice that we don't have direct access to that. We have to access the memory on the GPU 
   * by using these cuda function calls. We can't just reference that address because it's in 
   * a completely separate memory. It's kinda like we can't reference a memory on a different
   * processor when we're doing MPI, while we have to do some communication. In a sense, when 
   * we use these cuda function calls, to move information from a CPU memory to a GPU memory
   * and back, we're doing a similar kind of operation - we don't have direct access to that 
   * memory dans le sense de just putting an address out on the address bus of the motherboard
   * We have access to it by merit of making a function call.
   * So what's coming back from cudaMalloc here, is not a pointer as such.
   * It's sort of a memory reference that is only meaningful on GPU memory
   * cudaMalloc([address of a pointer, (void**) makes the compiler happy], memory bytes)
  */
  hipMalloc((void**)&d_in, arrayBytes);
  hipMalloc((void**)&d_out, arrayBytes);


  // Copy over - Compute - Copy back
  // copy array to GPU
  hipMemcpy(d_in, h_in, arrayBytes, hipMemcpyHostToDevice);
  /**
   * cudaMemcpy(
   * [reference that we have on the CPU side to memory that was actually allocated on the GPU],
   * [an address on the CPU side],
   * [how many bytes do we want to move from h_in to d_in],
   * [which direction we're going])
  */ 

  // launch the kernel
  /**
   * we want to spin it up on arraySize threads
   * we're gonna have one of those square functions running on each of 64 threads
   * (allocated one thread per element of the array)
  */
  square<<<1, arraySize>>>(d_out, d_in);

  // copy results back from GPU
  hipMemcpy(h_out, d_out, arrayBytes, hipMemcpyDeviceToHost);

  // Clean up
  // release GPU memory
  hipFree(d_in);
  hipFree(d_out);
}

